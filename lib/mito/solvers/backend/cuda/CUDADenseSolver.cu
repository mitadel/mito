// -*- c++ -*-
//
// Copyright (c) 2020-2024, the MiTo Authors, all rights reserved
//


#include "public.h"


// CUDA error checking macro
#define CHECK_CUDA_ERROR(call)                                                   \
    do {                                                                         \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            fprintf(stderr, "CUDA error at %s:%d\n", __FILE__, __LINE__);        \
            fprintf(stderr, "  Error: %s (%d)\n", hipGetErrorString(err), err); \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// function to convert cuSOLVER error codes to strings
inline const char *
cusolverGetErrorString(hipsolverStatus_t status)
{
    switch (status) {
        case HIPSOLVER_STATUS_SUCCESS:
            return "HIPSOLVER_STATUS_SUCCESS";
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            return "HIPSOLVER_STATUS_NOT_INITIALIZED";
        case HIPSOLVER_STATUS_ALLOC_FAILED:
            return "HIPSOLVER_STATUS_ALLOC_FAILED";
        case HIPSOLVER_STATUS_INVALID_VALUE:
            return "HIPSOLVER_STATUS_INVALID_VALUE";
        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            return "HIPSOLVER_STATUS_ARCH_MISMATCH";
        case HIPSOLVER_STATUS_MAPPING_ERROR:
            return "HIPSOLVER_STATUS_MAPPING_ERROR";
        case HIPSOLVER_STATUS_EXECUTION_FAILED:
            return "HIPSOLVER_STATUS_EXECUTION_FAILED";
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            return "HIPSOLVER_STATUS_INTERNAL_ERROR";
        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        default:
            return "Unknown cuSOLVER error";
    }
}

// cuSOLVER error checking macro
#define CHECK_CUSOLVER_ERROR(call)                                                                 \
    do {                                                                                           \
        hipsolverStatus_t status = (call);                                                          \
        if (status != HIPSOLVER_STATUS_SUCCESS) {                                                   \
            fprintf(                                                                               \
                stderr, "cuSOLVER error: %s at %s:%d\n", cusolverGetErrorString(status), __FILE__, \
                __LINE__);                                                                         \
            exit(EXIT_FAILURE);                                                                    \
        }                                                                                          \
    } while (0)

// constructor
template<mito::solvers::cuda::real_c realT>
mito::solvers::cuda::CUDADenseSolver<realT>::CUDADenseSolver(mito::solvers::cuda::SolverType solver_type) :
    _solver_type(solver_type),
    _h_matrix(nullptr),
    _h_rhs(nullptr),
    _h_solution(nullptr),
    _d_matrix(nullptr),
    _d_rhs(nullptr),
    _size(0),
    _is_solver_initialized(false),
    _allocated_host_memory_type(0),
    _is_assembly_finalized(false),
    _cusolver_handle(nullptr),
    _cuda_stream(nullptr)
{
    // initialize cuSOLVER
    _initialize_cusolver();
}

// destructor
template<mito::solvers::cuda::real_c realT>
mito::solvers::cuda::CUDADenseSolver<realT>::~CUDADenseSolver()
{
    // finalize cuSOLVER
    _finalize_cusolver();
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::initialize(size_t size) -> void
{
    // check if the solver is already initialized
    if (_is_solver_initialized) {
        throw std::logic_error(
            "Solver is already initialized. Are you sure you want to reinitialize? Then call "
            "finalize() first.");
    }

    // check if the size is valid
    if (size <= 0) {
        throw std::invalid_argument("Size of the linear system must be greater than zero.");
    }

    // save the size of the linear system
    _size = size;

    // allocate host memory
    _allocate_host_memory(size);

    // initialize host data
    _initialize_host_data(size);

    // allocate device memory
    _allocate_device_memory(size);

    // turn on the solver initialized flag
    _is_solver_initialized = true;

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::finalize() -> void
{
    // check if the solver is initialized
    if (_is_solver_initialized) {
        // free host memory
        _free_host_memory();

        // free device memory
        _free_device_memory();
    }

    // reset the solver initialized flag
    _is_solver_initialized = false;

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::reset_system() -> void
{
    // check if the solver is initialized
    if (!_is_solver_initialized) {
        throw std::logic_error("Solver is not yet initialized. Call initialize() first.");
    }

    // fill the host matrix, rhs and solution with zeros
    _initialize_host_data(_size);

    // reset the assembly finalized flag
    _is_assembly_finalized = false;

    // all done
    return;
}

// add/insert {value} to matrix entry at ({row}, {col}) of the host copy
template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::set_matrix_value(
    size_t row, size_t col, const real_type value,
    const mito::solvers::cuda::InsertMode insert_mode = mito::solvers::cuda::InsertMode::ADD_VALUE)
    -> void
{
    // check if the system assembly is finalized and throw an error if it is
    if (_is_assembly_finalized) {
        throw std::logic_error(
            "System assembly is already finalized. Cannot add/insert values to the matrix.");
    }

    // check if the row and column indices are within bounds
    _check_index_validity(row);
    _check_index_validity(col);

    // add/insert the value to the matrix entry in the host matrix
    // NOTE: We store the matrix in column-major order since the cuSOLVER library expects the matrix
    // to be in column-major order.
    if (insert_mode == mito::solvers::cuda::InsertMode::ADD_VALUE)
        _h_matrix[col * _size + row] += value;
    else if (insert_mode == mito::solvers::cuda::InsertMode::INSERT_VALUE)
        _h_matrix[col * _size + row] = value;
    else
        throw std::invalid_argument("Invalid insert mode. Use ADD_VALUE or INSERT_VALUE.");

    // all done
    return;
}

// add/insert {value} to rhs entry at {row} of the host copy
template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::set_rhs_value(
    size_t row, const real_type value,
    const mito::solvers::cuda::InsertMode insert_mode = mito::solvers::cuda::InsertMode::ADD_VALUE)
    -> void
{
    // check if the system assembly is finalized and throw an error if it is
    if (_is_assembly_finalized) {
        throw std::logic_error(
            "System assembly is already finalized. Cannot add/insert values to the rhs.");
    }

    // check if the row index is within bounds
    _check_index_validity(row);

    // add/insert the value to the rhs entry in the host rhs
    if (insert_mode == mito::solvers::cuda::InsertMode::ADD_VALUE)
        _h_rhs[row] += value;
    else if (insert_mode == mito::solvers::cuda::InsertMode::INSERT_VALUE)
        _h_rhs[row] = value;
    else
        throw std::invalid_argument("Invalid insert mode. Use ADD_VALUE or INSERT_VALUE.");

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::finalize_assembly() -> void
{
    // check if the solver is initialized
    if (!_is_solver_initialized) {
        throw std::logic_error(
            "Solver is not yet initialized. Call initialize() first, assemble the "
            "system, and then finalize the assembly.");
    }

    // issue a warning that all entries should be set before finalizing the assembly
    std::cerr
        << "Warning: Finalizing assembly. Make sure all system entries are set before this step."
        << std::endl;

    // set the assembly finalized flag to true
    _is_assembly_finalized = true;

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::solve() -> void
{
    // check if the assembly is finalized
    if (!_is_assembly_finalized) {
        throw std::logic_error(
            "System assembly is not yet finalized. Call finalize_assembly() first.");
    }

    // copy the host matrix and rhs data to device global memory
    // IMPROVE: We should move the data through streams for better performance later!
    CHECK_CUDA_ERROR(hipMemcpy(
        _d_matrix, _h_matrix, _size * _size * sizeof(real_type), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(
        hipMemcpy(_d_rhs, _h_rhs, _size * sizeof(real_type), hipMemcpyHostToDevice));

    // allocate device memory for temporary variables in the factorization
    int * d_pivot = nullptr;
    int * d_info = nullptr;
    real_type * d_workspace = nullptr;
    int workspace_size = 0;

    // check if real_type is either double or float and throw an error if it is not
    static_assert(
        std::is_same_v<real_type, double> || std::is_same_v<real_type, float>,
        "Only double or float types are supported in the CUDA dense solver.");

    // check the solver type is either LU or Cholesky
    if (_solver_type != mito::solvers::cuda::SolverType::LU
        && _solver_type != mito::solvers::cuda::SolverType::CHOLESKY) {
        throw std::invalid_argument(
            "Invalid solver type. Only LU and Cholesky solvers are supported in the CUDA dense "
            "solver.");
    }

    // get the workspace size for the factorization
    if (_solver_type == mito::solvers::cuda::SolverType::LU) {
        CHECK_CUSOLVER_ERROR(
            cusolver_traits<real_type>::getrf_buffer_size(
                _cusolver_handle, _size, _size, _d_matrix, _size, &workspace_size));
    } else if (_solver_type == mito::solvers::cuda::SolverType::CHOLESKY) {
        CHECK_CUSOLVER_ERROR(
            cusolver_traits<real_type>::potrf_buffer_size(
                _cusolver_handle, HIPBLAS_FILL_MODE_LOWER, _size, _d_matrix, _size,
                &workspace_size));
    }

    CHECK_CUDA_ERROR(hipMalloc(&d_pivot, _size * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_info, sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_workspace, workspace_size * sizeof(real_type)));

    // perform the factorization
    if (_solver_type == mito::solvers::cuda::SolverType::LU) {
        CHECK_CUSOLVER_ERROR(
            cusolver_traits<real_type>::getrf(
                _cusolver_handle, _size, _size, _d_matrix, _size, d_workspace, d_pivot, d_info));
    } else if (_solver_type == mito::solvers::cuda::SolverType::CHOLESKY) {
        CHECK_CUSOLVER_ERROR(
            cusolver_traits<real_type>::potrf(
                _cusolver_handle, HIPBLAS_FILL_MODE_LOWER, _size, _d_matrix, _size, d_workspace,
                workspace_size, d_info));
    }
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // solve the linear system
    if (_solver_type == mito::solvers::cuda::SolverType::LU) {
        CHECK_CUSOLVER_ERROR(
            cusolver_traits<real_type>::getrs(
                _cusolver_handle, HIPBLAS_OP_N, _size, 1, _d_matrix, _size, d_pivot, _d_rhs, _size,
                d_info));
    } else if (_solver_type == mito::solvers::cuda::SolverType::CHOLESKY) {
        CHECK_CUSOLVER_ERROR(
            cusolver_traits<real_type>::potrs(
                _cusolver_handle, HIPBLAS_FILL_MODE_LOWER, _size, 1, _d_matrix, _size, _d_rhs, _size,
                d_info));
    }
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // copy the solution from device global memory to host memory
    // NOTE: _d_rhs contains the solution after the call to getrs as its contents are overwritten
    // by the solution vector
    CHECK_CUDA_ERROR(
        hipMemcpy(_h_solution, _d_rhs, _size * sizeof(real_type), hipMemcpyDeviceToHost));

    // free the temporary device memory
    CHECK_CUDA_ERROR(hipFree(d_pivot));
    CHECK_CUDA_ERROR(hipFree(d_info));
    CHECK_CUDA_ERROR(hipFree(d_workspace));

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::_initialize_cusolver() -> void
{
    // create the cuSOLVER handle
    CHECK_CUSOLVER_ERROR(hipsolverDnCreate(&_cusolver_handle));

    // create a cuda stream
    CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&_cuda_stream, hipStreamNonBlocking, 0));

    // set the stream for the cuSOLVER handle
    CHECK_CUSOLVER_ERROR(hipsolverSetStream(_cusolver_handle, _cuda_stream));

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::_finalize_cusolver() -> void
{
    // destroy the cuSOLVER handle
    CHECK_CUSOLVER_ERROR(hipsolverDnDestroy(_cusolver_handle));

    // destroy the cuda stream
    CHECK_CUDA_ERROR(hipStreamDestroy(_cuda_stream));

    // reset the handle and stream pointers
    _cusolver_handle = nullptr;
    _cuda_stream = nullptr;

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::_allocate_host_memory(size_t size) -> void
{
    // try to allocate pinned memory on the host for faster transfers
    hipError_t err_pinned_alloc_matrix =
        hipHostMalloc(&_h_matrix, size * size * sizeof(real_type));
    hipError_t err_pinned_alloc_rhs = hipHostMalloc(&_h_rhs, size * sizeof(real_type));
    hipError_t err_pinned_alloc_solution = hipHostMalloc(&_h_solution, size * sizeof(real_type));

    // check if the pinned memory allocation for matrix, rhs, and solution was successful
    if (err_pinned_alloc_matrix == hipSuccess && err_pinned_alloc_rhs == hipSuccess
        && err_pinned_alloc_solution == hipSuccess) {
        // set the flag to indicate that pinned memory was allocated
        _allocated_host_memory_type = 1;
        return;
    }

    // free any partially allocated pinned memory
    if (err_pinned_alloc_matrix == hipSuccess)
        CHECK_CUDA_ERROR(hipHostFree(_h_matrix));
    if (err_pinned_alloc_rhs == hipSuccess)
        CHECK_CUDA_ERROR(hipHostFree(_h_rhs));
    if (err_pinned_alloc_solution == hipSuccess)
        CHECK_CUDA_ERROR(hipHostFree(_h_solution));

    // try to allocate regular memory on the host
    try {
        _h_matrix = new real_type[size * size];
        _h_rhs = new real_type[size];
        _h_solution = new real_type[size];
        // set the flag to indicate that regular memory was allocated
        _allocated_host_memory_type = 2;
    } catch (const std::bad_alloc & e) {
        throw std::runtime_error(
            "Failed to allocate host memory for matrix, rhs, and solution: "
            + std::string(e.what()));
    }

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::_allocate_device_memory(size_t size) -> void
{
    // allocate global device memory for matrix, rhs, and solution
    CHECK_CUDA_ERROR(hipMalloc(&_d_matrix, size * size * sizeof(real_type)));
    CHECK_CUDA_ERROR(hipMalloc(&_d_rhs, size * sizeof(real_type)));

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::_initialize_host_data(size_t size) -> void
{
    // check if host memory is allocated
    if (_allocated_host_memory_type == 0) {
        // throw developer error
        throw std::logic_error(
            "Host memory is not yet allocated. Call _allocate_host_memory() first.");
    }

    // initialize the host matrix, rhs and solution with zeros
    for (size_t i = 0; i < size * size; ++i) {
        _h_matrix[i] = 0.0;
    }
    for (size_t i = 0; i < size; ++i) {
        _h_rhs[i] = 0.0;
        _h_solution[i] = 0.0;
    }

    // all done
    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::_free_host_memory() -> void
{
    // check if pinned memory was allocated
    if (_allocated_host_memory_type == 1) {
        // free pinned memory
        if (_h_matrix)
            CHECK_CUDA_ERROR(hipHostFree(_h_matrix));
        if (_h_rhs)
            CHECK_CUDA_ERROR(hipHostFree(_h_rhs));
        if (_h_solution)
            CHECK_CUDA_ERROR(hipHostFree(_h_solution));
    } else if (_allocated_host_memory_type == 2) {
        // free regular memory
        if (_h_matrix)
            delete[] _h_matrix;
        if (_h_rhs)
            delete[] _h_rhs;
        if (_h_solution)
            delete[] _h_solution;
    }

    // reset the flag to indicate that the memory has been freed
    _allocated_host_memory_type = 0;
    // reset the pointers
    _h_matrix = nullptr;
    _h_rhs = nullptr;
    _h_solution = nullptr;

    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::_free_device_memory() -> void
{
    // free global device memory for matrix and rhs
    if (_d_matrix)
        CHECK_CUDA_ERROR(hipFree(_d_matrix));
    if (_d_rhs)
        CHECK_CUDA_ERROR(hipFree(_d_rhs));

    // reset the pointers
    _d_matrix = nullptr;
    _d_rhs = nullptr;

    return;
}

template<mito::solvers::cuda::real_c realT>
auto
mito::solvers::cuda::CUDADenseSolver<realT>::_check_index_validity(size_t index) const -> void
{
    // check if the solver is initialized
    // QUESTION: checking multiple times for initialization may be inefficient?
    if (!_is_solver_initialized) {
        throw std::logic_error("Solver is not yet initialized. Call initialize() first.");
    }

    // check if the index is valid and return false if it is not
    if (index >= _size) {
        throw std::out_of_range(
            "Index " + std::to_string(index) + " is out of range. It must be between 0 and "
            + std::to_string(_size - 1) + ".");
    }

    // all done
    return;
}

// explicit instantiation of the {CUDADenseSolver} class for doubles
template class mito::solvers::cuda::CUDADenseSolver<double>;

// explicit instantiation of the {CUDADenseSolver} class for floats
template class mito::solvers::cuda::CUDADenseSolver<float>;