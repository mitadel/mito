// -*- c++ -*-
//
// Copyright (c) 2020-2024, the MiTo Authors, all rights reserved
//


#include "public.h"


// CUDA error checking macro
#define CHECK_CUDA_ERROR(call)                                                   \
    do {                                                                         \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            fprintf(stderr, "CUDA error at %s:%d\n", __FILE__, __LINE__);        \
            fprintf(stderr, "  Error: %s (%d)\n", hipGetErrorString(err), err); \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// function to convert cuSOLVER error codes to strings
inline const char *
cusolverGetErrorString(hipsolverStatus_t status)
{
    switch (status) {
        case HIPSOLVER_STATUS_SUCCESS:
            return "HIPSOLVER_STATUS_SUCCESS";
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            return "HIPSOLVER_STATUS_NOT_INITIALIZED";
        case HIPSOLVER_STATUS_ALLOC_FAILED:
            return "HIPSOLVER_STATUS_ALLOC_FAILED";
        case HIPSOLVER_STATUS_INVALID_VALUE:
            return "HIPSOLVER_STATUS_INVALID_VALUE";
        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            return "HIPSOLVER_STATUS_ARCH_MISMATCH";
        case HIPSOLVER_STATUS_MAPPING_ERROR:
            return "HIPSOLVER_STATUS_MAPPING_ERROR";
        case HIPSOLVER_STATUS_EXECUTION_FAILED:
            return "HIPSOLVER_STATUS_EXECUTION_FAILED";
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            return "HIPSOLVER_STATUS_INTERNAL_ERROR";
        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        default:
            return "Unknown cuSOLVER error";
    }
}

// cuSOLVER error checking macro
#define CHECK_CUSOLVER_ERROR(call)                                                                 \
    do {                                                                                           \
        hipsolverStatus_t status = (call);                                                          \
        if (status != HIPSOLVER_STATUS_SUCCESS) {                                                   \
            fprintf(                                                                               \
                stderr, "cuSOLVER error: %s at %s:%d\n", cusolverGetErrorString(status), __FILE__, \
                __LINE__);                                                                         \
            exit(EXIT_FAILURE);                                                                    \
        }                                                                                          \
    } while (0)

// constructor
mito::solvers::cuda::CUDADenseSolver::CUDADenseSolver() :
    _h_matrix(nullptr),
    _h_rhs(nullptr),
    _h_solution(nullptr),
    _d_matrix(nullptr),
    _d_rhs(nullptr),
    _size(0),
    _is_solver_initialized(false),
    _allocated_host_memory_type(0),
    _is_assembly_finalized(false),
    _cusolver_handle(nullptr),
    _cuda_stream(nullptr)
{
    // initialize cuSOLVER
    _initialize_cusolver();
}

// destructor
mito::solvers::cuda::CUDADenseSolver::~CUDADenseSolver()
{
    // finalize cuSOLVER
    _finalize_cusolver();
}

auto
mito::solvers::cuda::CUDADenseSolver::initialize(size_t size) -> void
{
    // check if the solver is already initialized
    if (_is_solver_initialized) {
        throw std::logic_error(
            "Solver is already initialized. Are you sure you want to reinitialize? Then call "
            "finalize() first.");
    }

    // check if the size is valid
    if (size <= 0) {
        throw std::invalid_argument("Size of the linear system must be greater than zero.");
    }

    // save the size of the linear system
    _size = size;

    // allocate host memory
    _allocate_host_memory(size);

    // initialize host data
    _initialize_host_data(size);

    // allocate device memory
    _allocate_device_memory(size);

    // turn on the solver initialized flag
    _is_solver_initialized = true;

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::finalize() -> void
{
    // check if the solver is initialized
    if (_is_solver_initialized) {
        // free host memory
        _free_host_memory();

        // free device memory
        _free_device_memory();
    }

    // reset the solver initialized flag
    _is_solver_initialized = false;

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::reset_system() -> void
{
    // check if the solver is initialized
    if (!_is_solver_initialized) {
        throw std::logic_error("Solver is not yet initialized. Call initialize() first.");
    }

    // fill the host matrix, rhs and solution with zeros
    _initialize_host_data(_size);

    // reset the assembly finalized flag
    _is_assembly_finalized = false;

    // all done
    return;
}

// add/insert {value} to matrix entry at ({row}, {col}) of the host copy
auto
mito::solvers::cuda::CUDADenseSolver::set_matrix_value(
    size_t row, size_t col, const double value,
    const mito::solvers::cuda::InsertMode insert_mode = mito::solvers::cuda::InsertMode::ADD_VALUE)
    -> void
{
    // check if the system assembly is finalized and throw an error if it is
    if (_is_assembly_finalized) {
        throw std::logic_error(
            "System assembly is already finalized. Cannot add/insert values to the matrix.");
    }

    // check if the row and column indices are within bounds
    _check_index_validity(row);
    _check_index_validity(col);

    // add/insert the value to the matrix entry in the host matrix
    if (insert_mode == mito::solvers::cuda::InsertMode::ADD_VALUE)
        _h_matrix[row * _size + col] += value;
    else if (insert_mode == mito::solvers::cuda::InsertMode::INSERT_VALUE)
        _h_matrix[row * _size + col] = value;
    else
        throw std::invalid_argument("Invalid insert mode. Use ADD_VALUE or INSERT_VALUE.");

    // all done
    return;
}

// add/insert {value} to rhs entry at {row} of the host copy
auto
mito::solvers::cuda::CUDADenseSolver::set_rhs_value(
    size_t row, const double value,
    const mito::solvers::cuda::InsertMode insert_mode = mito::solvers::cuda::InsertMode::ADD_VALUE)
    -> void
{
    // check if the system assembly is finalized and throw an error if it is
    if (_is_assembly_finalized) {
        throw std::logic_error(
            "System assembly is already finalized. Cannot add/insert values to the rhs.");
    }

    // check if the row index is within bounds
    _check_index_validity(row);

    // add/insert the value to the rhs entry in the host rhs
    if (insert_mode == mito::solvers::cuda::InsertMode::ADD_VALUE)
        _h_rhs[row] += value;
    else if (insert_mode == mito::solvers::cuda::InsertMode::INSERT_VALUE)
        _h_rhs[row] = value;
    else
        throw std::invalid_argument("Invalid insert mode. Use ADD_VALUE or INSERT_VALUE.");

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::finalize_assembly() -> void
{
    // check if the solver is initialized
    if (!_is_solver_initialized) {
        throw std::logic_error(
            "Solver is not yet initialized. Call initialize() first, assemble the "
            "system, and then finalize the assembly.");
    }

    // issue a warning that all entries should be set before finalizing the assembly
    std::cerr
        << "Warning: Finalizing assembly. Make sure all system entries are set before this step."
        << std::endl;

    // set the assembly finalized flag to true
    _is_assembly_finalized = true;

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::solve() -> void
{
    // check if the assembly is finalized
    if (!_is_assembly_finalized) {
        throw std::logic_error(
            "System assembly is not yet finalized. Call finalize_assembly() first.");
    }

    // copy the host matrix and rhs data to device global memory
    // IMPROVE: We should move the data through streams for better performance later!
    CHECK_CUDA_ERROR(
        hipMemcpy(_d_matrix, _h_matrix, _size * _size * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(_d_rhs, _h_rhs, _size * sizeof(double), hipMemcpyHostToDevice));

    // allocate device memory for temporary variables in the LU factorization
    int * d_pivot = nullptr;
    int * d_info = nullptr;
    double * d_workspace = nullptr;
    int workspace_size = 0;

    // get the workspace size for getrf (only double precision LU factorization is supported!)
    // QUESTION: Should we check if mito::real is double or float and allocate the workspace memory
    // accordingly?
    CHECK_CUSOLVER_ERROR(hipsolverDnDgetrf_bufferSize(
        _cusolver_handle, _size, _size, _d_matrix, _size, &workspace_size));

    CHECK_CUDA_ERROR(hipMalloc(&d_pivot, _size * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_info, sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_workspace, workspace_size * sizeof(double)));

    // perform LU factorization
    CHECK_CUSOLVER_ERROR(hipsolverDnDgetrf(
        _cusolver_handle, _size, _size, _d_matrix, _size, d_workspace, d_pivot, d_info));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // solve the linear system
    CHECK_CUSOLVER_ERROR(hipsolverDnDgetrs(
        _cusolver_handle, HIPBLAS_OP_N, _size, 1, _d_matrix, _size, d_pivot, _d_rhs, _size, d_info));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // copy the solution from device global memory to host memory
    // NOTE: _d_rhs contains the solution after the call to getrs as its contents are overwritten
    // by the solution vector
    CHECK_CUDA_ERROR(
        hipMemcpy(_h_solution, _d_rhs, _size * sizeof(double), hipMemcpyDeviceToHost));

    // free the temporary device memory
    CHECK_CUDA_ERROR(hipFree(d_pivot));
    CHECK_CUDA_ERROR(hipFree(d_info));
    CHECK_CUDA_ERROR(hipFree(d_workspace));

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::_initialize_cusolver() -> void
{
    // create the cuSOLVER handle
    CHECK_CUSOLVER_ERROR(hipsolverDnCreate(&_cusolver_handle));

    // create a cuda stream
    CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&_cuda_stream, hipStreamNonBlocking, 0));

    // set the stream for the cuSOLVER handle
    CHECK_CUSOLVER_ERROR(hipsolverSetStream(_cusolver_handle, _cuda_stream));

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::_finalize_cusolver() -> void
{
    // destroy the cuSOLVER handle
    CHECK_CUSOLVER_ERROR(hipsolverDnDestroy(_cusolver_handle));

    // destroy the cuda stream
    CHECK_CUDA_ERROR(hipStreamDestroy(_cuda_stream));

    // reset the handle and stream pointers
    _cusolver_handle = nullptr;
    _cuda_stream = nullptr;

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::_allocate_host_memory(size_t size) -> void
{
    // try to allocate pinned memory on the host for faster transfers
    hipError_t err_pinned_alloc_matrix = hipHostMalloc(&_h_matrix, size * size * sizeof(double));
    hipError_t err_pinned_alloc_rhs = hipHostMalloc(&_h_rhs, size * sizeof(double));
    hipError_t err_pinned_alloc_solution = hipHostMalloc(&_h_solution, size * sizeof(double));

    // check if the pinned memory allocation for matrix, rhs, and solution was successful
    if (err_pinned_alloc_matrix == hipSuccess && err_pinned_alloc_rhs == hipSuccess
        && err_pinned_alloc_solution == hipSuccess) {
        // set the flag to indicate that pinned memory was allocated
        _allocated_host_memory_type = 1;
        return;
    }

    // free any partially allocated pinned memory
    if (err_pinned_alloc_matrix == hipSuccess)
        CHECK_CUDA_ERROR(hipHostFree(_h_matrix));
    if (err_pinned_alloc_rhs == hipSuccess)
        CHECK_CUDA_ERROR(hipHostFree(_h_rhs));
    if (err_pinned_alloc_solution == hipSuccess)
        CHECK_CUDA_ERROR(hipHostFree(_h_solution));

    // try to allocate regular memory on the host
    try {
        _h_matrix = new double[size * size];
        _h_rhs = new double[size];
        _h_solution = new double[size];
        // set the flag to indicate that regular memory was allocated
        _allocated_host_memory_type = 2;
    } catch (const std::bad_alloc & e) {
        throw std::runtime_error(
            "Failed to allocate host memory for matrix, rhs, and solution: "
            + std::string(e.what()));
    }

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::_allocate_device_memory(size_t size) -> void
{
    // allocate global device memory for matrix, rhs, and solution
    CHECK_CUDA_ERROR(hipMalloc(&_d_matrix, size * size * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&_d_rhs, size * sizeof(double)));

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::_initialize_host_data(size_t size) -> void
{
    // check if host memory is allocated
    if (_allocated_host_memory_type == 0) {
        // throw developer error
        throw std::logic_error(
            "Host memory is not yet allocated. Call _allocate_host_memory() first.");
    }

    // initialize the host matrix, rhs and solution with zeros
    for (size_t i = 0; i < size * size; ++i) {
        _h_matrix[i] = 0.0;
    }
    for (size_t i = 0; i < size; ++i) {
        _h_rhs[i] = 0.0;
        _h_solution[i] = 0.0;
    }

    // all done
    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::_free_host_memory() -> void
{
    // check if pinned memory was allocated
    if (_allocated_host_memory_type == 1) {
        // free pinned memory
        if (_h_matrix)
            CHECK_CUDA_ERROR(hipHostFree(_h_matrix));
        if (_h_rhs)
            CHECK_CUDA_ERROR(hipHostFree(_h_rhs));
        if (_h_solution)
            CHECK_CUDA_ERROR(hipHostFree(_h_solution));
    } else if (_allocated_host_memory_type == 2) {
        // free regular memory
        if (_h_matrix)
            delete[] _h_matrix;
        if (_h_rhs)
            delete[] _h_rhs;
        if (_h_solution)
            delete[] _h_solution;
    }

    // reset the flag to indicate that the memory has been freed
    _allocated_host_memory_type = 0;
    // reset the pointers
    _h_matrix = nullptr;
    _h_rhs = nullptr;
    _h_solution = nullptr;

    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::_free_device_memory() -> void
{
    // free global device memory for matrix and rhs
    if (_d_matrix)
        CHECK_CUDA_ERROR(hipFree(_d_matrix));
    if (_d_rhs)
        CHECK_CUDA_ERROR(hipFree(_d_rhs));

    // reset the pointers
    _d_matrix = nullptr;
    _d_rhs = nullptr;

    return;
}

auto
mito::solvers::cuda::CUDADenseSolver::_check_index_validity(size_t index) const -> void
{
    // check if the solver is initialized
    // QUESTION: checking multiple times for initialization may be inefficient?
    if (!_is_solver_initialized) {
        throw std::logic_error("Solver is not yet initialized. Call initialize() first.");
    }

    // check if the index is valid and return false if it is not
    if (index >= _size) {
        throw std::out_of_range(
            "Index " + std::to_string(index) + " is out of range. It must be between 0 and "
            + std::to_string(_size - 1) + ".");
    }

    // all done
    return;
}